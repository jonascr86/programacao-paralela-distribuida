#include "hip/hip_runtime.h"


#include <stdio.h>
#include <math.h>

__global__ void adicionarKernel(double* resultado, const double* n) {
	int i = threadIdx.x;
	double a = 1, b = 0;
	double delta = pow(b, 2) - (4 * a * (n[i] * -1));
	resultado[i] = ((b * -1) + sqrt(delta)) / 2 * a;
}

hipError_t calcularRaizes(double* resultado, const double* n, unsigned int tam) {
	double* gpu_n;
	double* gpu_resultado;
	hipError_t status;

	status = hipSetDevice(0);
	if (status != hipSuccess) {
		printf("hipSetDevice falhou!");
		goto Falha;
	}

	status = hipMalloc((void**)&gpu_resultado, tam * sizeof(double));
	if (status != hipSuccess) {
		printf("hipMalloc falhou!");
		goto Falha;
	}

	status = hipMalloc((void**)&gpu_n, tam * sizeof(double));
	if (status != hipSuccess) {
		printf("hipMalloc falhou!");
		goto Falha;
	}

	status = hipMemcpy(gpu_n, n, tam * sizeof(double), hipMemcpyHostToDevice);
	if (status != hipSuccess) {
		printf("hipMemcpy falhou!");
		goto Falha;
	}

	adicionarKernel<<<1, tam>>>(gpu_resultado, gpu_n);

	status = hipGetLastError();
	if (status != hipSuccess) {
		printf("adicionarKernel falhou: %s", hipGetErrorString(status));
		goto Falha;
	}

	status = hipDeviceSynchronize();
	if (status != hipSuccess) {
		printf("hipDeviceSynchronize falhou: %d", status);
		goto Falha;
	}

	status = hipMemcpy(resultado, gpu_resultado, tam * sizeof(double), hipMemcpyDeviceToHost);
	if (status != hipSuccess) {
		printf("hipMemcpy falhou!");
		goto Falha;
	}

Falha:
	hipFree(gpu_resultado);
	hipFree(gpu_n);

	return status;
}

int main()
{
	const int tam = 10;
	const double n[tam] = { 1, 4, 9, 16, 25, 36, 49, 64, 81, 100 };
	double resultado[tam] = { 0 };

	hipError_t status = calcularRaizes(resultado, n, tam);
	if (status != hipSuccess) {
		printf("calcularRaizes falhou!");
		return 1;
;	}

	printf("RESULTADO\n");
	for (int i = 0; i < tam; i++) {
		printf("Raiz de %f = %f\n", n[i], resultado[i]);
	}

	getchar();

	status = hipDeviceReset();
	if (status != hipSuccess) {
		printf("hipDeviceReset falhou!");
		return 1;
	}

	return 0;
}